#include "model.hpp"
#include "huffman.hpp"
#include "unigram.hpp"
#include <iostream>
#include <fstream>
#include <sstream>
#include <cinttypes>
#include <queue>
#include <hip/hip_runtime.h>
using llu = uint64_t;
using llf = double;

extern int debug;

// could be parallelized
void file_to_docs(const Vocab& vocab, std::string train_file, size_t *&docs) {
    if (debug > 1) {
        std::cout << "Converting train file to indices." << std::endl;
    }
    std::vector<size_t> tmp;
    std::fstream fs(train_file);
    std::string ln;
    const size_t eol = vocab.get_id("</s>");
    while (std::getline(fs, ln)) {
        std::stringstream ss(ln);
        std::string w;
        while (ss >> w) {
            if (vocab.contain(w)) {
                tmp.push_back(vocab.get_id(w));
            }
        }
        tmp.push_back(eol);
    }
    if (debug > 1) {
        std::cout << "Total Size: " << tmp.size() * sizeof(size_t) / 1024 << " KiB" << std::endl;
    }
    hipMallocManaged(&docs, tmp.size() * sizeof(size_t));
    memcpy(docs, tmp.data(), tmp.size() * sizeof(size_t));
}

void init_net(llf *&syn0, llf *&syn1, llf *&syn1neg, const ModelConfig& conf, size_t vocab_size) {
    hipMallocManaged(&syn0, vocab_size * conf.layer_size * sizeof(llf));
    llu rnd = 1;
    for (size_t i = 0; i < vocab_size; ++i) {
        for (size_t j = 0; j < conf.layer_size; ++j) {
            rnd = rnd * 25214903917LLU + 11;
            syn0[i * conf.layer_size + j] = (static_cast<llf>(rnd & 0xffff) / 0xffff - 0.5) / conf.layer_size;
        }
    }

    if (conf.hierarchical_softmax) {
        hipMallocManaged(&syn1, vocab_size * conf.layer_size * sizeof(llf));
        hipMemset(syn1, 0, vocab_size * conf.layer_size * sizeof(llf));
    }

    if (conf.negative_sample > 0) {
        hipMallocManaged(&syn1neg, vocab_size * conf.layer_size * sizeof(llf));
        hipMemset(syn1neg, 0, vocab_size * conf.layer_size * sizeof(llf));
    }
}

void train_model(const Vocab& vocab, const ModelConfig& conf) {
    VocabWord *words;
    build_binary_tree(vocab, words);

    size_t *docs;
    file_to_docs(vocab, conf.train_file, docs);

    size_t *unigram;
    if (conf.negative_sample > 0) {
        init_unigram_table(vocab, unigram);
    }

    llf *syn0, *syn1, *syn1neg;
    init_net(syn0, syn1, syn1neg, conf, vocab.size());

    if (conf.cbow) {
        if (conf.hierarchical_softmax) {
        } else {
        }
    } else {
        if (conf.hierarchical_softmax) {
        } else {
        }
    }
}
